
#include "global.h"
#include "hip/hip_runtime.h"


namespace cuda {

	__host__
	void check_error() {
		hipError_t error = hipGetLastError();
		if (error != hipSuccess) {
			throw std::runtime_error(hipGetErrorString(error));
		}
	}

	__host__
	int optimize_threads_void(const void * func, size_t shared_size_static, size_t shared_size_per_thread) {

		// TODO: device selection

		int max_block_size;
		hipDeviceGetAttribute(&max_block_size, hipDeviceAttributeMaxThreadsPerBlock, 0);
		check_error();

		int best_block_size = 0;
		for (int block_size=1; block_size<=max_block_size; block_size*=2) {
			int num_blocks;
			size_t shared_size = shared_size_static + shared_size_per_thread*block_size;
			hipOccupancyMaxActiveBlocksPerMultiprocessor(&num_blocks, func, block_size, shared_size);
			check_error();
			if (num_blocks <= 0) {
				break;
			} else {
				best_block_size = block_size;
			}
		}

		return best_block_size;
	}
}
