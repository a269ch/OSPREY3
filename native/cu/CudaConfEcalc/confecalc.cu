#include "hip/hip_runtime.h"

#include "global.h"
#include "hip/hip_runtime.h"
#include "array.h"
#include "formats.h"
#include "rotation.h"
#include "atoms.h"
#include "confspace.h"
#include "assignment.h"
#include "energy.h"
#include "energy_ambereef1.h"
/* TEMP
#include "motions.h"
#include "minimization.h"
*/

#define API extern "C" [[maybe_unused]]


API int version_major() {
	return CudaConfEcalc_VERSION_MAJOR;
}

API int version_minor() {
	return CudaConfEcalc_VERSION_MINOR;
}

API int cuda_version_driver() {
	int v;
	hipDriverGetVersion(&v);
	return v;
}

API int cuda_version_runtime() {
	int v;
	hipRuntimeGetVersion(&v);
	switch (hipGetLastError()) {
		case hipSuccess: break;
		case hipErrorInsufficientDriver: return -1;
		case hipErrorNoDevice: return -2;
		default: return INT_MIN;
	}
	return v;
}

API int cuda_version_required() {
	return CUDART_VERSION;
}

template<typename T>
static void * alloc_conf_space(const osprey::ConfSpace<T> & conf_space) {

	// allocate the device memory
	void * p_device;
	hipMalloc(&p_device, conf_space.size);
	cuda::check_error();

	// upload the conf space
	hipMemcpy(p_device, &conf_space, conf_space.size, hipMemcpyHostToDevice);
	cuda::check_error();

	return p_device;
}

API void * alloc_conf_space_f32(const osprey::ConfSpace<float32_t> & conf_space) {
	return alloc_conf_space(conf_space);
}
API void * alloc_conf_space_f64(const osprey::ConfSpace<float64_t> & conf_space) {
	return alloc_conf_space(conf_space);
}

API void free_conf_space(void * p) {
	hipFree(p);
	cuda::check_error();
}


template<typename T>
__global__
void assign_kernel(const osprey::ConfSpace<T> * conf_space, const osprey::Array<int32_t> * conf, osprey::Array<osprey::Real3<T>> * out_coords) {

	cg::thread_block threads = cg::this_thread_block();

	// slice up the shared memory
	extern __shared__ int8_t shared[];
	int64_t shared_offset = 0;
	auto shared_atom_pairs = reinterpret_cast<const void **>(shared + shared_offset);
	shared_offset += osprey::Assignment<T>::sizeof_atom_pairs(conf_space->num_pos);
	auto shared_conf_energies = reinterpret_cast<T *>(shared + shared_offset);

	// init the sizes for device-allocated Array instances
	out_coords->init(conf_space->max_num_conf_atoms, threads);

	osprey::Assignment<T> assignment(*conf_space, *conf, *out_coords, shared_atom_pairs, shared_conf_energies, threads);
}

template<typename T>
static void assign(const osprey::ConfSpace<T> * d_conf_space,
                   const osprey::Array<int32_t> & conf,
                   osprey::Array<osprey::Real3<T>> & out_coords) {

	// upload the arguments
	size_t conf_bytes = conf.get_bytes();
	osprey::Array<int32_t> * d_conf;
	hipMalloc(&d_conf, conf_bytes);
	cuda::check_error();
	hipMemcpy(d_conf, &conf, conf_bytes, hipMemcpyHostToDevice);
	cuda::check_error();

	// allocate space for the coords
	size_t out_coords_bytes = out_coords.get_bytes();
	osprey::Array<osprey::Real3<T>> * d_out_coords;
	hipMalloc(&d_out_coords, out_coords_bytes);
	cuda::check_error();

	// compute the shared memory size
	int64_t shared_size = 0
		+ osprey::Assignment<T>::sizeof_atom_pairs(conf.get_size())
		+ osprey::Assignment<T>::sizeof_conf_energies(conf.get_size());

	// launch the kernel
	int num_threads = cuda::optimize_threads(assign_kernel<T>, shared_size);
	assign_kernel<<<1, num_threads, shared_size>>>(d_conf_space, d_conf, d_out_coords);
	cuda::check_error();

	// download the coords
	hipMemcpy(&out_coords, d_out_coords, out_coords_bytes, hipMemcpyDeviceToHost);
	cuda::check_error();

	// cleanup
	hipFree(d_conf);
	hipFree(d_out_coords);
}

API void assign_f32(const osprey::ConfSpace<float32_t> * d_conf_space,
                    const osprey::Array<int32_t> & conf,
                    osprey::Array<osprey::Real3<float32_t>> & out_coords) {
	assign(d_conf_space, conf, out_coords);
}
API void assign_f64(const osprey::ConfSpace<float64_t> * d_conf_space,
                    const osprey::Array<int32_t> & conf,
                    osprey::Array<osprey::Real3<float64_t>> & out_coords) {
	assign(d_conf_space, conf, out_coords);
}


template<typename T>
using CalcKernel = void (*)(const osprey::ConfSpace<T> *,
                            const osprey::Array<int32_t> *,
                            const osprey::Array<osprey::PosInter<T>> *,
                            osprey::Array<osprey::Real3<T>> *,
                            T *);

template<typename T>
__global__
void calc_kernel_amber_eef1(const osprey::ConfSpace<T> * conf_space,
                            const osprey::Array<int32_t> * conf,
                            const osprey::Array<osprey::PosInter<T>> * inters,
                            osprey::Array<osprey::Real3<T>> * out_coords,
                            T * out_energy) {

	cg::thread_block threads = cg::this_thread_block();

	// slice up the shared memory
	extern __shared__ int8_t shared[];
	int64_t shared_offset = 0;
	auto shared_atom_pairs = reinterpret_cast<const void **>(shared + shared_offset);
	shared_offset += osprey::Assignment<T>::sizeof_atom_pairs(conf_space->num_pos);
	auto shared_conf_energies = reinterpret_cast<T *>(shared + shared_offset);

	// init the sizes for device-allocated Array instances
	out_coords->init(conf_space->max_num_conf_atoms, threads);

	// make the atoms
	osprey::Assignment<T> assignment(*conf_space, *conf, *out_coords, shared_atom_pairs, shared_conf_energies, threads);

	// call the energy function
	*out_energy = osprey::ambereef1::calc_energy(assignment, *inters);
}

template<typename T>
static T calc(const osprey::ConfSpace<T> * d_conf_space,
              const osprey::Array<int32_t> & conf,
              const osprey::Array<osprey::PosInter<T>> & inters,
              osprey::Array<osprey::Real3<T>> * out_coords,
              int64_t num_atoms,
              CalcKernel<T> kernel) {

	// upload the arguments
	size_t conf_bytes = conf.get_bytes();
	osprey::Array<int32_t> * d_conf;
	hipMalloc(&d_conf, conf_bytes);
	cuda::check_error();
	hipMemcpy(d_conf, &conf, conf_bytes, hipMemcpyHostToDevice);
	cuda::check_error();

	size_t inters_bytes = inters.get_bytes();
	osprey::Array<osprey::PosInter<T>> * d_inters;
	hipMalloc(&d_inters, inters_bytes);
	cuda::check_error();
	hipMemcpy(d_inters, &inters, inters_bytes, hipMemcpyHostToDevice);
	cuda::check_error();

	// TODO: combine allocations/transfers for speed?

	// allocate space for the coords
	size_t out_coords_bytes = osprey::Array<osprey::Real3<T>>::get_bytes(num_atoms);
	osprey::Array<osprey::Real3<T>> * d_out_coords;
	hipMalloc(&d_out_coords, out_coords_bytes);
	cuda::check_error();

	// allocate space for the energy
	size_t out_energy_bytes = sizeof(T);
	T * d_out_energy;
	hipMalloc(&d_out_energy, out_energy_bytes);
	cuda::check_error();

	// compute the shared memory size
	int64_t shared_size = 0
		+ osprey::Assignment<T>::sizeof_atom_pairs(conf.get_size())
		+ osprey::Assignment<T>::sizeof_conf_energies(conf.get_size());

	// launch the kernel
	// TODO: cache thread optimization
	int num_threads = cuda::optimize_threads(*kernel, shared_size);
	kernel<<<1, num_threads, shared_size>>>(d_conf_space, d_conf, d_inters, d_out_coords, d_out_energy);
	cuda::check_error();

	// download the energy
	T out_energy;
	hipMemcpy(&out_energy, d_out_energy, out_energy_bytes, hipMemcpyDeviceToHost);
	cuda::check_error();

	// download the coords, if needed
	if (out_coords != nullptr) {
		hipMemcpy(out_coords, d_out_coords, out_coords_bytes, hipMemcpyDeviceToHost);
		cuda::check_error();
	}

	// cleanup
	hipFree(d_conf);
	hipFree(d_inters);
	hipFree(d_out_coords);
	hipFree(d_out_energy);

	return out_energy;
}

API float32_t calc_amber_eef1_f32(const osprey::ConfSpace<float32_t> * d_conf_space,
                                  const osprey::Array<int32_t> & conf,
                                  const osprey::Array<osprey::PosInter<float32_t>> & inters,
                                  osprey::Array<osprey::Real3<float32_t>> * out_coords,
                                  int64_t num_atoms) {
	return calc<float32_t>(d_conf_space, conf, inters, out_coords, num_atoms, calc_kernel_amber_eef1);
}
API float64_t calc_amber_eef1_f64(const osprey::ConfSpace<float64_t> * d_conf_space,
								  const osprey::Array<int32_t> & conf,
                                  const osprey::Array<osprey::PosInter<float64_t>> & inters,
                                  osprey::Array<osprey::Real3<float64_t>> * out_coords,
                                  int64_t num_atoms) {
	return calc<float64_t>(d_conf_space, conf, inters, out_coords, num_atoms, calc_kernel_amber_eef1);
}


/* TODO

template<typename T>
static T minimize(const osprey::ConfSpace<T> & conf_space, const int32_t conf[],
                  const osprey::Array<osprey::PosInter<T>> & inters,
                  osprey::EnergyFunction<T> efunc,
                  osprey::Array<osprey::Real3<T>> * out_coords, osprey::Array<T> * out_dofs) {

	// make the coords and the degrees of freedom
	osprey::Assignment<T> assignment(conf_space, conf);
	osprey::Dofs<T> dofs(assignment, inters, efunc);

	// init the dofs to the center of the voxel
	osprey::DofValues<T> vals(dofs.get_size());
	for (int d=0; d<dofs.get_size(); d++) {
		vals.x[d] = dofs[d].center();
	}

	// use CCD to do the minimization
	osprey::minimize_ccd(dofs, vals);

	// set out the results
	if (out_coords != nullptr) {
		out_coords->copy_from(assignment.atoms);
	}
	if (out_dofs != nullptr) {
		out_dofs->copy_from(vals.x, dofs.get_size());
		out_dofs->truncate(dofs.get_size());
	}
	return vals.f;
}

API float32_t minimize_amber_eef1_f32(const osprey::ConfSpace<float32_t> & conf_space, const int32_t conf[],
                                      const osprey::Array<osprey::PosInter<float32_t>> & inters,
                                      osprey::Array<osprey::Real3<float32_t>> * out_coords, osprey::Array<float32_t> * out_dofs) {
	return minimize<float32_t>(conf_space, conf, inters, osprey::ambereef1::calc_energy, out_coords, out_dofs);
}
API float64_t minimize_amber_eef1_f64(const osprey::ConfSpace<float64_t> & conf_space, const int32_t conf[],
                                      const osprey::Array<osprey::PosInter<float64_t>> & inters,
                                      osprey::Array<osprey::Real3<float64_t>> * out_coords, osprey::Array<float64_t> * out_dofs) {
	return minimize<float64_t>(conf_space, conf, inters, osprey::ambereef1::calc_energy, out_coords, out_dofs);
}

*/
