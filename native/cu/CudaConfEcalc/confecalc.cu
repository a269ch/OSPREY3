#include "hip/hip_runtime.h"

#include "global.h"
#include "hip/hip_runtime.h"
#include "array.h"
#include "formats.h"
#include "rotation.h"
#include "atoms.h"
#include "confspace.h"
#include "assignment.h"
#include "energy.h"
#include "energy_ambereef1.h"
#include "motions.h"
#include "minimization.h"

#define API extern "C" [[maybe_unused]]


API int version_major() {
	return CudaConfEcalc_VERSION_MAJOR;
}

API int version_minor() {
	return CudaConfEcalc_VERSION_MINOR;
}

API int cuda_version_driver() {
	int v;
	hipDriverGetVersion(&v);
	return v;
}

API int cuda_version_runtime() {
	int v;
	hipRuntimeGetVersion(&v);
	switch (hipGetLastError()) {
		case hipSuccess: break;
		case hipErrorInsufficientDriver: return -1;
		case hipErrorNoDevice: return -2;
		default: return INT_MIN;
	}
	return v;
}

API int cuda_version_required() {
	return CUDART_VERSION;
}

struct GpuInfo {
	char bus_id[16];
	char name[256];
	int integrated;
	int concurrent_kernels;
	int num_processors;
	int num_async_engines;
	int64_t mem_total;
	int64_t mem_free;
};
ASSERT_JAVA_COMPATIBLE(GpuInfo, 304);

API osprey::Array<GpuInfo> * alloc_gpu_infos() {

	int num_gpus;
	CUDACHECK(hipGetDeviceCount(&num_gpus));

	osprey::Array<GpuInfo> * infos = osprey::Array<GpuInfo>::make(num_gpus);
	for (int device=0; device<num_gpus; device++) {
		GpuInfo & info = (*infos)[device];

		CUDACHECK(hipDeviceGetPCIBusId(reinterpret_cast<char *>(&info.bus_id), 16, device));

		hipDeviceProp_t props {};
		CUDACHECK(hipGetDeviceProperties(&props, device));

		memcpy(&info.name, props.name, 256);
		info.integrated = props.integrated;
		info.concurrent_kernels = props.concurrentKernels;
		info.num_processors = props.multiProcessorCount;
		info.num_async_engines = props.asyncEngineCount;

		size_t mem_free;
		size_t mem_total;
		CUDACHECK(hipMemGetInfo(&mem_free, &mem_total));

		info.mem_free = mem_free;
		info.mem_total = mem_total;
	}

	return infos;
}

API void free_gpu_infos(osprey::Array<GpuInfo> * p) {
	delete[] p;
}


namespace osprey {

	template<typename T>
	static void * alloc_conf_space(int device, const ConfSpace<T> & conf_space) {

		CUDACHECK(hipSetDevice(device));

		// allocate the device memory
		void * p_device;
		CUDACHECK(hipMalloc(&p_device, conf_space.size));

		// upload the conf space
		CUDACHECK(hipMemcpy(p_device, &conf_space, conf_space.size, hipMemcpyHostToDevice));

		return p_device;
	}
}

API void * alloc_conf_space_f32(int device, const osprey::ConfSpace<float32_t> & conf_space) {
	return osprey::alloc_conf_space(device, conf_space);
}
API void * alloc_conf_space_f64(int device, const osprey::ConfSpace<float64_t> & conf_space) {
	return osprey::alloc_conf_space(device, conf_space);
}

API void free_conf_space(int device, void * p) {

	CUDACHECK(hipSetDevice(device));

	CUDACHECK(hipFree(p));
}


class Stream {
	public:

		static const size_t d_buf_size = 2*1024*1024; // 8 MiB, big enough to hold all the coords
		static const size_t h_buf_size = 16; // just big enough to hold the resulting energy

		__host__
		inline Stream() {
			CUDACHECK(hipStreamCreate(&stream));
			CUDACHECK(hipHostMalloc(&h_buf, h_buf_size));
			CUDACHECK(hipMalloc(&d_buf, d_buf_size));
		}

		Stream(const Stream & other) = delete;

		__host__
		inline ~Stream() {
			hipHostFree(h_buf);
			hipFree(d_buf);
			hipStreamDestroy(stream);
		}

		inline hipStream_t get_stream() const {
			return stream;
		}

		inline int8_t * get_h_buf() {
			return h_buf;
		}

		inline int8_t * get_d_buf() {
			return d_buf;
		}

	private:
		hipStream_t stream;
		int8_t * h_buf;
		int8_t * d_buf;
};

API Stream * alloc_stream(int device) {
	CUDACHECK(hipSetDevice(device));
	return new Stream();
}

API void free_stream(int device, Stream * stream) {
	CUDACHECK(hipSetDevice(device));
	delete stream;
}


namespace osprey {

	template<typename T>
	__global__
	void assign_kernel(const ConfSpace<T> * conf_space, const Array<int32_t> * conf, Array<osprey::Real3<T>> * out_coords) {

		// slice up the shared memory
		extern __shared__ int8_t shared[];
		int64_t shared_offset = 0;
		auto shared_atom_pairs = reinterpret_cast<const void **>(shared + shared_offset);
		shared_offset += Assignment<T>::sizeof_atom_pairs(conf_space->num_pos);
		auto shared_conf_energies = reinterpret_cast<T *>(shared + shared_offset);

		// init the sizes for hipMalloc'd Array instances
		out_coords->init(conf_space->max_num_conf_atoms);

		Assignment<T> assignment(*conf_space, *conf, *out_coords, shared_atom_pairs, shared_conf_energies);
	}

	template<typename T>
	static void assign(int device,
	                   Stream * stream,
	                   const ConfSpace<T> * d_conf_space,
	                   const Array<int32_t> & conf,
	                   Array<Real3<T>> & out_coords) {

		CUDACHECK(hipSetDevice(device));

		// upload the arguments
		size_t conf_bytes = conf.get_bytes();
		Array<int32_t> * d_conf;
		CUDACHECK(hipMalloc(&d_conf, conf_bytes));
		CUDACHECK(hipMemcpy(d_conf, &conf, conf_bytes, hipMemcpyHostToDevice));

		// allocate space for the coords
		size_t out_coords_bytes = out_coords.get_bytes();
		Array<Real3<T>> * d_out_coords;
		CUDACHECK(hipMalloc(&d_out_coords, out_coords_bytes));

		// compute the shared memory size
		int64_t shared_size = 0
			+ Assignment<T>::sizeof_atom_pairs(conf.get_size())
			+ Assignment<T>::sizeof_conf_energies(conf.get_size());

		// launch the kernel
		// TODO: optimize launch bounds
		int num_threads = cuda::optimize_threads(assign_kernel<T>, shared_size, 0);
		assign_kernel<<<1, num_threads, shared_size, stream->get_stream()>>>(d_conf_space, d_conf, d_out_coords);
		cuda::check_error();

		// download the coords
		CUDACHECK(hipMemcpy(&out_coords, d_out_coords, out_coords_bytes, hipMemcpyDeviceToHost));

		// cleanup
		CUDACHECK(hipFree(d_conf));
		CUDACHECK(hipFree(d_out_coords));
	}
}

API void assign_f32(int device,
                    Stream * stream,
                    const osprey::ConfSpace<float32_t> * d_conf_space,
                    const osprey::Array<int32_t> & conf,
                    osprey::Array<osprey::Real3<float32_t>> & out_coords) {
	osprey::assign(device, stream, d_conf_space, conf, out_coords);
}
API void assign_f64(int device,
                    Stream * stream,
                    const osprey::ConfSpace<float64_t> * d_conf_space,
                    const osprey::Array<int32_t> & conf,
                    osprey::Array<osprey::Real3<float64_t>> & out_coords) {
	osprey::assign(device, stream, d_conf_space, conf, out_coords);
}


namespace osprey {

	#define CALC_THREADS 1024
	#define CALC_BLOCKS 1

	template<typename T, osprey::EnergyFunction<T> efunc>
	__global__
	__launch_bounds__(CALC_THREADS, CALC_BLOCKS)
	void calc_kernel(const osprey::ConfSpace<T> * conf_space,
	                 const osprey::Array<int32_t> * conf,
	                 const osprey::Array<osprey::PosInter<T>> * inters,
	                 osprey::Array<osprey::Real3<T>> * out_coords,
	                 T * out_energy) {

		// slice up the shared memory
		extern __shared__ int8_t _shared[];
		auto shared = _shared;

		auto shared_atom_pairs = reinterpret_cast<const void **>(shared);
		assert (cuda::is_aligned<16>(shared_atom_pairs));
		shared += cuda::pad_to_alignment<16>(Assignment<T>::sizeof_atom_pairs(conf_space->num_pos));

		auto shared_conf_energies = reinterpret_cast<T *>(shared);
		assert (cuda::is_aligned<16>(shared_conf_energies));
		shared += cuda::pad_to_alignment<16>(Assignment<T>::sizeof_conf_energies(conf_space->num_pos));

		auto thread_energy = reinterpret_cast<T *>(shared);
		assert (cuda::is_aligned<16>(shared_conf_energies));

		// init the sizes for hipMalloc'd Array instances
		out_coords->init(conf_space->max_num_conf_atoms);

		// make the atoms
		osprey::Assignment<T> assignment(*conf_space, *conf, *out_coords, shared_atom_pairs, shared_conf_energies);

		// call the energy function
		*out_energy = efunc(assignment, *inters, thread_energy);
	}

	template<typename T, osprey::EnergyFunction<T> efunc>
	static T calc(int device,
	              Stream * stream,
	              const osprey::ConfSpace<T> * d_conf_space,
	              const osprey::Array<int32_t> & conf,
	              const osprey::Array<osprey::PosInter<T>> & inters,
	              osprey::Array<osprey::Real3<T>> * out_coords,
	              int64_t num_atoms) {

		CUDACHECK(hipSetDevice(device));

		// upload the arguments
		size_t conf_bytes = conf.get_bytes();
		osprey::Array<int32_t> * d_conf;
		CUDACHECK(hipMalloc(&d_conf, conf_bytes));
		CUDACHECK(hipMemcpy(d_conf, &conf, conf_bytes, hipMemcpyHostToDevice));

		size_t inters_bytes = inters.get_bytes();
		osprey::Array<osprey::PosInter<T>> * d_inters;
		CUDACHECK(hipMalloc(&d_inters, inters_bytes));
		CUDACHECK(hipMemcpy(d_inters, &inters, inters_bytes, hipMemcpyHostToDevice));

		// TODO: combine allocations/transfers for speed?

		// allocate space for the coords
		size_t out_coords_bytes = osprey::Array<osprey::Real3<T>>::get_bytes(num_atoms);
		osprey::Array<osprey::Real3<T>> * d_out_coords;
		CUDACHECK(hipMalloc(&d_out_coords, out_coords_bytes));

		// allocate space for the energy
		size_t out_energy_bytes = sizeof(T);
		T * d_out_energy;
		CUDACHECK(hipMalloc(&d_out_energy, out_energy_bytes));

		// compute the shared memory size
		int64_t shared_size_static = 0
			+ cuda::pad_to_alignment<16>(Assignment<T>::sizeof_atom_pairs(conf.get_size()))
			+ cuda::pad_to_alignment<16>(Assignment<T>::sizeof_conf_energies(conf.get_size()));
		int64_t shared_size_per_thread = sizeof(T);

		// launch the kernel
		int num_threads = CALC_THREADS;
		int64_t shared_size = shared_size_static + shared_size_per_thread*num_threads;
		calc_kernel<T,efunc><<<1, num_threads, shared_size, stream->get_stream()>>>(d_conf_space, d_conf, d_inters, d_out_coords, d_out_energy);
		cuda::check_error();

		// download the energy
		T out_energy;
		CUDACHECK(hipMemcpy(&out_energy, d_out_energy, out_energy_bytes, hipMemcpyDeviceToHost));

		// download the coords, if needed
		if (out_coords != nullptr) {
			CUDACHECK(hipMemcpy(out_coords, d_out_coords, out_coords_bytes, hipMemcpyDeviceToHost));
		}

		// cleanup
		CUDACHECK(hipFree(d_conf));
		CUDACHECK(hipFree(d_inters));
		CUDACHECK(hipFree(d_out_coords));
		CUDACHECK(hipFree(d_out_energy));

		return out_energy;
	}
}

API float32_t calc_amber_eef1_f32(int device,
                                  Stream * stream,
                                  const osprey::ConfSpace<float32_t> * d_conf_space,
                                  const osprey::Array<int32_t> & conf,
                                  const osprey::Array<osprey::PosInter<float32_t>> & inters,
                                  osprey::Array<osprey::Real3<float32_t>> * out_coords,
                                  int64_t num_atoms) {
	return osprey::calc<float32_t,osprey::ambereef1::calc_energy>(device, stream, d_conf_space, conf, inters, out_coords, num_atoms);
}
API float64_t calc_amber_eef1_f64(int device,
                                  Stream * stream,
                                  const osprey::ConfSpace<float64_t> * d_conf_space,
                                  const osprey::Array<int32_t> & conf,
                                  const osprey::Array<osprey::PosInter<float64_t>> & inters,
                                  osprey::Array<osprey::Real3<float64_t>> * out_coords,
                                  int64_t num_atoms) {
	return osprey::calc<float64_t,osprey::ambereef1::calc_energy>(device, stream, d_conf_space, conf, inters, out_coords, num_atoms);
}


namespace osprey {

	// configure the launch bounds, so the compiler knows how many registers to use
	#define MINIMIZE_THREADS_F32_VOLTA 1024
	#define MINIMIZE_THREADS_F64_VOLTA 1024
	#define MINIMIZE_THREADS_F32_PASCAL 1024 // TODO: optimize for pascal?
	#define MINIMIZE_THREADS_F64_PASCAL 1024
	#define MINIMIZE_THREADS_F32_MAXWELL 1024 // TODO: optimize for maxwell?
	#define MINIMIZE_THREADS_F64_MAXWELL 1024

	// the kernels use __syncthreads a lot, so they're designed to take up an entire SM
	#define MINIMIZE_BLOCKS 1

	#if __CUDA_ARCH__ >= 700 // volta
		#define MINIMIZE_THREADS_F32 MINIMIZE_THREADS_F32_VOLTA
		#define MINIMIZE_THREADS_F64 MINIMIZE_THREADS_F64_VOLTA
	#elif __CUDA_ARCH__ >= 600 // pascal
		#define MINIMIZE_THREADS_F32 MINIMIZE_THREADS_F32_PASCAL
		#define MINIMIZE_THREADS_F64 MINIMIZE_THREADS_F64_PASCAL
	#elif __CUDA_ARCH__ >= 500 // maxwell
		#define MINIMIZE_THREADS_F32 MINIMIZE_THREADS_F32_MAXWELL
		#define MINIMIZE_THREADS_F64 MINIMIZE_THREADS_F64_MAXWELL
	#else
		// host side
		#define MINIMIZE_THREADS_F32 -1
		#define MINIMIZE_THREADS_F64 -1
	#endif

	template<typename T>
	__host__
	inline int get_minimize_threads();

	template<>
	__host__
	inline int get_minimize_threads<float32_t>() {
		int arch = cuda::get_arch();
		if (arch >= 700) {
			return MINIMIZE_THREADS_F32_VOLTA;
		} else if (arch >= 600) {
			return MINIMIZE_THREADS_F32_PASCAL;
		} else if (arch >= 500) {
			return MINIMIZE_THREADS_F32_MAXWELL;
		} else {
			throw std::runtime_error("unsupported CUDA architecture");
		}
	}

	template<>
	__host__
	inline int get_minimize_threads<float64_t>() {
		int arch = cuda::get_arch();
		if (arch >= 700) {
			return MINIMIZE_THREADS_F64_VOLTA;
		} else if (arch >= 600) {
			return MINIMIZE_THREADS_F64_PASCAL;
		} else if (arch >= 500) {
			return MINIMIZE_THREADS_F64_MAXWELL;
		} else {
			throw std::runtime_error("unsupported CUDA architecture");
		}
	}

	template<typename T, EnergyFunction<T> efunc>
	__device__
	void minimize_kernel_impl(int64_t shared_size,
	                          const ConfSpace<T> * conf_space,
	                          const Array<int32_t> * conf,
	                          const Array<PosInter<T>> * inters,
	                          Array<Real3<T>> * out_coords,
	                          DofValues<T> * out_dof_values) {

		// slice up the shared memory
		extern __shared__ int8_t shared[];
		int64_t offset = 0;

		auto shared_atom_pairs = reinterpret_cast<const void **>(shared + offset);
		assert (cuda::is_aligned<16>(shared_atom_pairs));
		offset += cuda::pad_to_alignment<16>(Assignment<T>::sizeof_atom_pairs(conf_space->num_pos));

		auto shared_conf_energies = reinterpret_cast<T *>(shared + offset);
		assert (cuda::is_aligned<16>(shared_conf_energies));
		offset += cuda::pad_to_alignment<16>(Assignment<T>::sizeof_conf_energies(conf_space->num_pos));

		auto shared_dofs_mem = reinterpret_cast<int8_t *>(shared + offset);
		assert (cuda::is_aligned<16>(shared_dofs_mem));
		offset += Dofs<T>::dofs_shared_size;

		auto shared_dof_bufs = reinterpret_cast<int8_t *>(shared + offset);
		assert (cuda::is_aligned<16>(shared_dof_bufs));
		offset += (Dof<T>::buf_size + Array<const PosInter<T> *>::get_bytes(inters->get_size()))*conf_space->max_num_dofs;

		auto shared_line_search_states = reinterpret_cast<LineSearchState<T> *>(shared + offset);
		assert (cuda::is_aligned<16>(shared_line_search_states));
		offset += cuda::pad_to_alignment<16>(sizeof(LineSearchState<T>)*conf_space->max_num_dofs);

		auto shared_here = reinterpret_cast<DofValues<T> *>(shared + offset);
		assert (cuda::is_aligned<16>(shared_here));
		offset += DofValues<T>::get_bytes(conf_space->max_num_dofs);

		auto shared_next = reinterpret_cast<DofValues<T> *>(shared + offset);
		assert (cuda::is_aligned<16>(shared_next));
		offset += DofValues<T>::get_bytes(conf_space->max_num_dofs);

		auto thread_energy = reinterpret_cast<T *>(shared + offset);
		assert (cuda::is_aligned<16>(thread_energy));
		offset += sizeof(T)*blockDim.x;

		assert (offset == shared_size);

		// init the sizes for hipMalloc'd Array instances
		out_coords->init(conf_space->max_num_conf_atoms);
		out_dof_values->x.init(conf_space->max_num_dofs);

		// init sizes for Array instances in shared memory
		shared_here->x.init(conf_space->max_num_dofs);
		shared_next->x.init(conf_space->max_num_dofs);

		// make the atoms and the degrees of freedom
		Assignment<T> assignment(*conf_space, *conf, *out_coords, shared_atom_pairs, shared_conf_energies);
		Dofs<T> dofs(assignment, *inters, efunc, thread_energy, shared_dof_bufs, shared_dofs_mem);

		// truncate the dof values to match the actual number of dofs
		if (threadIdx.x == 0) {
			out_dof_values->x.truncate(dofs.get_size());
			shared_here->x.truncate(dofs.get_size());
			shared_next->x.truncate(dofs.get_size());
		}
		__syncthreads();

		// init the dofs to the center of the voxel
		for (int d=threadIdx.x; d<dofs.get_size(); d+=blockDim.x) {
			shared_here->x[d] = dofs[d].center();
		}
		__syncthreads();

		// use CCD to do the minimization
		minimize_ccd(dofs, *shared_here, *shared_next, shared_line_search_states);

		// copy dof values to global memory
		out_dof_values->set(*shared_here);
	}

	// set kernel launch bounds for each real type

	template<typename T, EnergyFunction<T> efunc>
	__global__
	void minimize_kernel(int64_t shared_size,
	                     const ConfSpace<T> * conf_space,
	                     const Array<int32_t> * conf,
	                     const Array<PosInter<T>> * inters,
	                     Array<Real3<T>> * out_coords,
	                     DofValues<T> * out_dof_values);

	template<>
	__global__
	__launch_bounds__(MINIMIZE_THREADS_F32, MINIMIZE_BLOCKS)
	void minimize_kernel<float32_t,osprey::ambereef1::calc_energy>(int64_t shared_size,
	                                                               const ConfSpace<float32_t> * conf_space,
	                                                               const Array<int32_t> * conf,
	                                                               const Array<PosInter<float32_t>> * inters,
	                                                               Array<Real3<float32_t>> * out_coords,
	                                                               DofValues<float32_t> * out_dof_values) {
		minimize_kernel_impl<float32_t,osprey::ambereef1::calc_energy>(shared_size, conf_space, conf, inters, out_coords, out_dof_values);
	}

	template<>
	__global__
	__launch_bounds__(MINIMIZE_THREADS_F64, MINIMIZE_BLOCKS)
	void minimize_kernel<float64_t,osprey::ambereef1::calc_energy>(int64_t shared_size,
	                                                               const ConfSpace<float64_t> * conf_space,
	                                                               const Array<int32_t> * conf,
	                                                               const Array<PosInter<float64_t>> * inters,
	                                                               Array<Real3<float64_t>> * out_coords,
	                                                               DofValues<float64_t> * out_dof_values) {
		minimize_kernel_impl<float64_t,osprey::ambereef1::calc_energy>(shared_size, conf_space, conf, inters, out_coords, out_dof_values);
	}

	template<typename T, EnergyFunction<T> efunc>
	static T minimize(int device,
	                  Stream * stream,
	                  const ConfSpace<T> * d_conf_space,
	                  const Array<int32_t> & conf,
	                  const Array<PosInter<T>> & inters,
	                  Array<Real3<T>> * out_coords,
	                  int64_t num_atoms,
	                  Array<T> * out_dofs,
	                  int64_t max_num_dofs) {

		CUDACHECK(hipSetDevice(device));

		// split up the transfer buffer
		size_t conf_bytes = conf.get_bytes();
		size_t inters_bytes = inters.get_bytes();
		size_t out_coords_bytes = Array<Real3<T>>::get_bytes(num_atoms);
		size_t out_dof_values_bytes = DofValues<T>::get_bytes(max_num_dofs);
		if (conf_bytes + inters_bytes + out_coords_bytes + out_dof_values_bytes > Stream::d_buf_size) {
			throw std::runtime_error("exceeded transfer buffer size");
		}

		int8_t * d_xfer_buf = stream->get_d_buf();

		auto d_conf = reinterpret_cast<Array<int32_t> *>(d_xfer_buf);
		assert (cuda::is_aligned<16>(d_conf));
		d_xfer_buf += conf_bytes;

		auto d_inters = reinterpret_cast<Array<PosInter<T>> *>(d_xfer_buf);
		assert (cuda::is_aligned<16>(d_inters));
		d_xfer_buf += inters_bytes;

		auto d_out_coords = reinterpret_cast<Array<Real3<T>> *>(d_xfer_buf);
		assert (cuda::is_aligned<16>(d_out_coords));
		d_xfer_buf += out_coords_bytes;

		auto d_out_dof_values = reinterpret_cast<DofValues<T> *>(d_xfer_buf);
		assert (cuda::is_aligned<16>(d_out_dof_values));

		// upload the arguments
		CUDACHECK(hipMemcpyAsync(d_conf, &conf, conf_bytes, hipMemcpyHostToDevice, stream->get_stream()));
		CUDACHECK(hipMemcpyAsync(d_inters, &inters, inters_bytes, hipMemcpyHostToDevice, stream->get_stream()));

		// compute the shared memory size
		int64_t shared_size_static = 0
			+ cuda::pad_to_alignment<16>(Assignment<T>::sizeof_atom_pairs(conf.get_size()))
			+ cuda::pad_to_alignment<16>(Assignment<T>::sizeof_conf_energies(conf.get_size()))
			+ (Dof<T>::buf_size + Array<const PosInter<T> *>::get_bytes(inters.get_size()))*max_num_dofs
			+ Dofs<T>::dofs_shared_size
			+ cuda::pad_to_alignment<16>(sizeof(LineSearchState<T>)*max_num_dofs) // line_search_states
			+ DofValues<T>::get_bytes(max_num_dofs) // here
			+ DofValues<T>::get_bytes(max_num_dofs); // next
		int64_t shared_size_per_thread = sizeof(T);

		// launch the kernel
		int num_threads = get_minimize_threads<T>();
		int64_t shared_size = shared_size_static + shared_size_per_thread*num_threads;
		minimize_kernel<T,efunc><<<1, num_threads, shared_size, stream->get_stream()>>>(shared_size, d_conf_space, d_conf, d_inters, d_out_coords, d_out_dof_values);
		cuda::check_error();

		// download the energy
		CUDACHECK(hipMemcpyAsync(stream->get_h_buf(), DofValues<T>::f_ptr(d_out_dof_values), sizeof(T), hipMemcpyDeviceToHost, stream->get_stream()));

		// download the coords, if needed
		if (out_coords != nullptr) {
			CUDACHECK(hipMemcpyAsync(out_coords, d_out_coords, out_coords_bytes, hipMemcpyDeviceToHost, stream->get_stream()));
		}

		// download the dofs, if needed
		if (out_dofs != nullptr) {
			// TODO: does this work?
			CUDACHECK(hipMemcpyAsync(out_dofs, DofValues<T>::x_ptr(d_out_dof_values), Array<T>::get_bytes(max_num_dofs), hipMemcpyDeviceToHost, stream->get_stream()));
		}

		// wait for the downloads to finish
		CUDACHECK(hipStreamSynchronize(stream->get_stream()));

		// finally, return the minimized energy
		const T * out_energy = reinterpret_cast<const T *>(stream->get_h_buf());
		return *out_energy;
	}
}

API float32_t minimize_amber_eef1_f32(int device,
                                      Stream * stream,
                                      const osprey::ConfSpace<float32_t> * d_conf_space,
                                      const osprey::Array<int32_t> & conf,
                                      const osprey::Array<osprey::PosInter<float32_t>> & inters,
                                      osprey::Array<osprey::Real3<float32_t>> * out_coords,
                                      int64_t num_atoms,
                                      osprey::Array<float32_t> * out_dofs,
                                      int64_t max_num_dofs) {
	return osprey::minimize<float32_t,osprey::ambereef1::calc_energy>(device, stream, d_conf_space, conf, inters, out_coords, num_atoms, out_dofs, max_num_dofs);
}
API float64_t minimize_amber_eef1_f64(int device,
                                      Stream * stream,
                                      const osprey::ConfSpace<float64_t> * d_conf_space,
                                      const osprey::Array<int32_t> & conf,
                                      const osprey::Array<osprey::PosInter<float64_t>> & inters,
                                      osprey::Array<osprey::Real3<float64_t>> * out_coords,
                                      int64_t num_atoms,
                                      osprey::Array<float64_t> * out_dofs,
                                      int64_t max_num_dofs) {
	return osprey::minimize<float64_t,osprey::ambereef1::calc_energy>(device, stream, d_conf_space, conf, inters, out_coords, num_atoms, out_dofs, max_num_dofs);
}
