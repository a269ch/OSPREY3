#include "hip/hip_runtime.h"

#include "global.h"
#include "hip/hip_runtime.h"
#include "array.h"
#include "formats.h"
#include "rotation.h"
#include "atoms.h"
#include "confspace.h"
#include "assignment.h"
#include "energy.h"
#include "energy_ambereef1.h"
#include "motions.h"
#include "minimization.h"
#include "confecalc.h"


namespace osprey {

	void exception_handler() {

		// get the stack trace
		const int SIZE = 32;
		void * trace_elems[SIZE];
		int num_frames = backtrace(trace_elems, SIZE);
		char ** symbols = backtrace_symbols(trace_elems, num_frames);

		// print it to the console
		std::cerr << "Unhandled exception in CUDA conformation energy calculator native code!" << std::endl;
		for (int i=0 ; i<num_frames; i++) {
			std::cerr << "\t" << symbols[i] << std::endl;
		}

		// cleanup
		free(symbols);

		// exit the program with an error code
		std::abort();
	}

	void free_conf_space(int device, void * p) {
		CUDACHECK(hipSetDevice(device));
		CUDACHECK(hipFree(p));
	}

	Stream * alloc_stream(int device, int64_t host_bytes, int64_t device_bytes) {
		CUDACHECK(hipSetDevice(device));
		return new Stream(host_bytes, device_bytes);
	}

	void free_stream(int device, Stream * stream) {
		CUDACHECK(hipSetDevice(device));
		delete stream;
	}


	// set kernel launch bounds for each real type

	template<>
	__global__
	__launch_bounds__(MINIMIZE_THREADS_F32, MINIMIZE_BLOCKS)
	void minimize_kernel<float32_t,osprey::ambereef1::calc_energy>(int64_t shared_size,
	                                                               const ConfSpace<float32_t> * conf_space,
	                                                               int64_t max_num_inters,
	                                                               int8_t * xfer_buf) {
		minimize_kernel_impl<float32_t,osprey::ambereef1::calc_energy>(shared_size, conf_space, max_num_inters, xfer_buf);
	}

	template<>
	__global__
	__launch_bounds__(MINIMIZE_THREADS_F64, MINIMIZE_BLOCKS)
	void minimize_kernel<float64_t,osprey::ambereef1::calc_energy>(int64_t shared_size,
	                                                               const ConfSpace<float64_t> * conf_space,
	                                                               int64_t max_num_inters,
	                                                               int8_t * xfer_buf) {
		minimize_kernel_impl<float64_t,osprey::ambereef1::calc_energy>(shared_size, conf_space, max_num_inters, xfer_buf);
	}
}
